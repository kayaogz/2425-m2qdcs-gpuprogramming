#include <cstdio>
#include <iostream>
#include "hip/hip_runtime.h"

using namespace std;

__global__ void cudaCopieParBlocs(float *tab0, const float *tab1, int taille)
{
  int idx = blockIdx.x;
  if (idx < taille) { tab0[idx] = tab1[idx]; }
}

__global__ void cudaCopieParBlocsThreads(float *tab0, const float *tab1, int taille)
{
  // Calculer le bon idx en fonction du blockIdx.x, threadIdx.x, et blockDim.x
  // A FAIRE
  int idx; // idx = ?
  idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < taille) { tab0[idx] = tab1[idx]; }
}

int main(int argc, char **argv) {
  float *A, *B, *Ad, *Bd;
  int N, i;

  if (argc < 2) {
    printf("Utilisation: ./cuda-copie N\n");
    return 0;
  }
  N = atoi(argv[1]);

  // Initialisation
  A = (float *) malloc(sizeof(float) * N);
  B = (float *) malloc(sizeof(float) * N);
  for (i = 0; i < N; i++) { 
    A[i] = (float)i;
    B[i] = 0.0f;
  }
  
  // Allouer les tableau Ad et Bd dynamiques de taille N sur le GPU avec cudaMalloc 
  // A FAIRE
  hipMalloc(&Ad, N * sizeof(float));
  hipMalloc(&Bd, N * sizeof(float));

  // Copier A dans Ad et B dans Bd
  // A FAIRE
  hipMemcpy(Ad, A, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Bd, B, N * sizeof(float), hipMemcpyHostToDevice);

  // Copier Ad dans Bd avec le kernel cudaCopieParBlocs
  // A FAIRE ...
  cudaCopieParBlocs<<<N, 1>>>(Bd, Ad, N);

  // Attendre que le kernel cudaCopieParBlocs termine
  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess) {
    printf("L'execution du kernel a echoue avec le code d'erreur \"%s\".\n", hipGetErrorString(cudaerr));
  }

  // Verifier le resultat en CPU en copiant Bd dans B puis en comparant B avec A
  // A FAIRE
  hipMemcpy(B, Bd, N * sizeof(float), hipMemcpyDeviceToHost);
  for (i = 0; i < N; i++) { if (A[i] != B[i]) { break; } }
  if (i < N) { cout << "La copie est incorrecte!\n"; }
  else { cout << "La copie est correcte!\n"; }

  // Remettre B et Bd a zero pour tester le deuxieme kernel de copie
  // A FAIRE
  for (int i = 0; i < N; i++) { B[i] = 0.0f; }
  hipMemcpy(Bd, B, N * sizeof(float), hipMemcpyHostToDevice);

  // Copier Ad dans Bd avec le kernel cudaCopieParBlocsThreads
  // A FAIRE ...
  cudaCopieParBlocsThreads<<<(N-1)/1024 + 1, 1024>>>(Bd, Ad, N);

  // Attendre que le kernel cudaCopieParBlocsThreads termine
  cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess) {
    printf("L'execution du kernel a echoue avec le code d'erreur \"%s\".\n", hipGetErrorString(cudaerr));
  }

  // Verifier le resultat en CPU en copiant Bd dans B puis en comparant B avec A
  // A FAIRE
  hipMemcpy(B, Bd, N * sizeof(float), hipMemcpyDeviceToHost);
  for (i = 0; i < N; i++) { if (A[i] != B[i]) { break; } }
  if (i < N) { cout << "La copie est incorrecte!\n"; }
  else { cout << "La copie est correcte!\n"; }

  // Desaollouer le tableau Ad[N] et Bd[N] sur le GPU
  // A FAIRE ...
  hipFree(Ad);
  hipFree(Bd);

  // Desallouer A et B
  free(A);
  free(B);

  return 0;
  }
