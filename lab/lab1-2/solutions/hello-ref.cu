#include <cstdio>
#include "hip/hip_runtime.h"

__global__ void cudaHello(){
  // Afficher le message Hello World ainsi que blockidx et threadidx depuis chaque thread
  // A FAIRE ...
  printf("Hello World from block %d, thread %d!\n", blockIdx.x, threadIdx.x);
}

int main() {
  int numBlocks = 64;
  int blockSize = 1;
  cudaHello<<<numBlocks, blockSize>>>(); 

  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess)
    printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
  return 0;
  }
