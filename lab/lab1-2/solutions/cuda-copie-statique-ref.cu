#include <cstdio>
#include <iostream>
#include "hip/hip_runtime.h"

using namespace std;

#define N 1024

// Definir un tableau de float Ad[N] de taille statique sur le GPU
// A FAIRE ...
__device__ float Ad[N];

int main() {
  float A[N], B[N];
  int i;

  // Initialisation
  for (i = 0; i < N; i++) { A[i] = (float)i; }

  // cudaMemcpy de A[N] vers Ad[N]
  // A FAIRE ...
  hipMemcpyToSymbol(HIP_SYMBOL(Ad), A, sizeof(float) * N, 0, hipMemcpyHostToDevice);

  // cudaMemcpy de Ad[N] vers B[N]
  // A FAIRE ...
  hipMemcpyFromSymbol(B, HIP_SYMBOL(Ad), sizeof(float) * N, 0, hipMemcpyDeviceToHost);

  // Attendre que les kernels GPUs terminent
  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess) {
    printf("L'execution du kernel a echoue avec le code d'erreur \"%s\".\n", hipGetErrorString(cudaerr));
  }

  // Verifier le resultat
  for (i = 0; i < N; i++) { if (A[i] != B[i]) { break; } }
  if (i < N) { cout << "La copie est incorrecte!\n"; }
  else { cout << "La copie est correcte!\n"; }

  return 0;
}
